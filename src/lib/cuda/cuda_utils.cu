#include "cuda_utils.cuh"

#include <sstream>

void cuda_check(const std::string &file, int line)
{
    static std::string prev_file;
    static int prev_line = 0;

    hipError_t e = hipGetLastError();
    if (e != hipSuccess)
    {
        std::stringstream ss;
        ss << file << ", line " << line << ": " << hipGetErrorString(e) << " (" << e << ")\n";
        if (prev_line > 0)
            ss << "Previous CUDA call:\n" << prev_file << ", line " << prev_line << "\n";
        throw std::runtime_error(ss.str());
    }

    prev_file = file;
    prev_line = line;
}
